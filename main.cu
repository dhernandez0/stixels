#include "hip/hip_runtime.h"
/**
    This file is part of stixels. (https://github.com/dhernandez0/stixels).

    Copyright (c) 2016 Daniel Hernandez Juarez.

    stixels is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    stixels is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with stixels.  If not, see <http://www.gnu.org/licenses/>.

**/

#include <iostream>
#include <fstream>
#include <dirent.h>
#include <sys/stat.h>
#include "Stixels.hpp"
#include "RoadEstimation.h"
#include "configuration.h"

#define OVERWRITE	true

void SaveStixels(std::vector<Section> *stixels, const int real_cols, const char *fname) {
	std::ofstream fp;
	fp.open (fname, std::ofstream::out | std::ofstream::trunc);
	//fp << "Writing this to a file.\n";
	if(fp.is_open()) {
		for(size_t i = 0; i < real_cols; i++) {
			std::vector<Section> sections_vec = stixels[i];
			for(size_t j = 0; j < sections_vec.size(); j++) {
				Section section = sections_vec.at(j);
				fp << section.type << "," << section.vB << "," << section.vT << "," << section.disparity << ";";
			}
			// Column finished
			fp << std::endl;
		}
		fp.close();
	} else {
		std::cerr << "Counldn't write file: " << fname << std::endl;
	}
}

void SaveStixels(Section *stixels, const int real_cols, const int max_segments, const char *fname) {
	std::ofstream fp;
	fp.open (fname, std::ofstream::out | std::ofstream::trunc);
	//fp << "Writing this to a file.\n";
	if(fp.is_open()) {
		for(size_t i = 0; i < real_cols; i++) {
			for(size_t j = 0; j < max_segments; j++) {
				Section section = stixels[i*max_segments+j];
				if(section.type == -1) {
					break;
				}
				// If disparity is 0 it is sky
				if(section.type == OBJECT && section.disparity < 1.0f) {
					section.type = SKY;
				}
				fp << section.type << "," << section.vB << "," << section.vT << "," << section.disparity << ";";
			}
			// Column finished
			fp << std::endl;
		}
		fp.close();
	} else {
		std::cerr << "Counldn't write file: " << fname << std::endl;
	}
}

bool FileExists(const char *fname) {
	struct stat buffer;
	return (stat (fname, &buffer) == 0);
}

int main(int argc, char *argv[]) {
	if(argc < 3) {
		std::cerr << "Usage: stixels dir max_disparity" << std::endl;
		return -1;
	}
	//nvtxNameOsThread("Stixels");
	const char* directory = argv[1];
	const int max_dis = atoi(argv[2]);
	const char* disparity_dir = "disparities";
	const char* stixel_dir = "stixels";

	DIR *dp;
	struct dirent *ep;
	char abs_dis_dir[PATH_MAX];
	sprintf(abs_dis_dir, "%s/%s", directory, disparity_dir);
	dp = opendir(abs_dis_dir);
	if (dp == NULL) {
		std::cerr << "Invalid directory: " << abs_dis_dir << std::endl;
		exit(EXIT_FAILURE);
	}
	char dis_file[PATH_MAX];
	char stixel_file[PATH_MAX];

	/* Parameters
	 *
	 */

	/* Disparity Parameters */
	const float sigma_disparity_object = 1.0f;
	const float sigma_disparity_ground = 2.0f;
	const float sigma_sky = 0.1f; // Should be small compared to sigma_dis

	/* Probabilities */
	const float pout = 0.15f;
	const float pout_sky = 0.4f;
	const float pord = 0.2f;
	const float pgrav = 0.1f;
	const float pblg = 0.04f;

	//
	// Must add 1
	const float pground_given_nexist = 0.36f;
	const float pobject_given_nexist = 0.28f;
	const float psky_given_nexist = 0.36f;

	const float pnexist_dis = 0.0f;
	const float pground = 1.0f/3.0f;
	const float pobject = 1.0f/3.0f;
	const float psky = 1.0f/3.0f;

	/* Camera Paramters */
	int vhor;

	// Virtual parameters
	const float focal = 704.7082f;
	const float baseline = 0.8f;
	const float camera_center_y = 384.0f;
	const int column_step = 5;
	const int width_margin = 0;

	float camera_tilt;
	const float sigma_camera_tilt = 0.05f;
	float camera_height;
	const float sigma_camera_height = 0.05f;
	//const float camera_center_x = 651.216186523f;
	float alpha_ground;

	/* Model Parameters */
	const bool median_step = false;
	const float epsilon = 3.0f;
	const float range_objects_z = 10.20f; // in meters

	bool first_time = true;
	Stixels stixles;
	RoadEstimation road_estimation;
	std::vector<float> times;
	pixel_t *im;

	while ((ep = readdir(dp)) != NULL) {
		if (!strcmp (ep->d_name, "."))
			continue;
		if (!strcmp (ep->d_name, ".."))
			continue;
		sprintf(dis_file, "%s/%s/%s", directory, disparity_dir, ep->d_name);
		sprintf(stixel_file, "%s/%s/%s.%s", directory, stixel_dir, ep->d_name, "stixels");

		if(!FileExists(stixel_file) || OVERWRITE) {
			cv::Mat dis = cv::imread(dis_file, cv::IMREAD_UNCHANGED);
			if(!dis.data) {
				std::cerr << "Couldn't read the file " << dis_file << std::endl;
				return EXIT_FAILURE;
			}

			// Convert images to grayscale
			if (dis.channels()>1) {
				cv::cvtColor(dis, dis, cv::COLOR_RGB2GRAY);
			}

			std::cout << ep->d_name << std::endl;

			const int rows = dis.rows;
			const int cols = dis.cols;

			if(rows < max_dis) {
				printf("ERROR: Image height has to be equal or bigger than maximum disparity\n");
				first_time = false;
				continue;
			}

			if(rows >= 1024) {
				printf("ERROR: Maximum image height has to be less than 1024\n");
				first_time = false;
				continue;
			}

			if(first_time) {
				stixles.SetDisparityParameters(rows, cols, max_dis, sigma_disparity_object, sigma_disparity_ground, sigma_sky);
				stixles.SetProbabilities(pout, pout_sky, pground_given_nexist, pobject_given_nexist, psky_given_nexist, pnexist_dis, pground, pobject, psky, pord, pgrav, pblg);
				stixles.SetModelParameters(column_step, median_step, epsilon, range_objects_z, width_margin);
				stixles.SetCameraParameters(0.0f, focal, baseline, 0.0f, sigma_camera_tilt, 0.0f, sigma_camera_height, 0.0f);
				stixles.Initialize();
				road_estimation.Initialize(camera_center_y, baseline, focal, rows, cols, max_dis);

				CUDA_CHECK_RETURN(hipHostMalloc((void**)&im, rows*cols*sizeof(pixel_t)));
			}
			if(dis.depth() == CV_8U) {
				for(int i = 0; i < dis.rows; i++) {
					for(int j = 0; j < dis.cols; j++) {
						const pixel_t d = (float) dis.at<uint8_t>(i, j);
						im[i*dis.cols+j] = d;
					}
				}
			} else {
				for(int i = 0; i < dis.rows; i++) {
					for(int j = 0; j < dis.cols; j++) {
						const pixel_t d = (float) dis.at<uint16_t>(i, j)/256.0f;
						im[i*dis.cols+j] = d;
					}
				}
			}

			// Compute some camera parameters
			stixles.SetDisparityImage(im);

			const bool ok = road_estimation.Compute(im);
			if(!ok) {
				printf("Can't compute road estimation\n");
				first_time = false;
				continue;
			}

			// Get Camera Parameters
			camera_tilt = road_estimation.GetPitch();
			camera_height = road_estimation.GetCameraHeight();
			vhor = road_estimation.GetHorizonPoint();
			alpha_ground = road_estimation.GetSlope();

			if(camera_tilt == 0 && camera_height == 0 && vhor == 0 && alpha_ground == 0) {
				printf("Can't compute road estimation\n");
				first_time = false;
				continue;
			}

			std::cout << "Camera Parameters -> Tilt: " << camera_tilt << " Height: " << camera_height << " vHor: " << vhor << " alpha_ground: " << alpha_ground << std::endl;

			stixles.SetCameraParameters(vhor, focal, baseline, camera_tilt, sigma_camera_tilt, camera_height, sigma_camera_height, alpha_ground);

			const float elapsed_time_ms = stixles.Compute();
			times.push_back(elapsed_time_ms);

			Section *stx = stixles.GetStixels();

			SaveStixels(stx, stixles.GetRealCols(), stixles.GetMaxSections(), stixel_file);
			first_time = false;
		}
	}
	if(!first_time) {
		stixles.Finish();
		road_estimation.Finish();
	}

	float mean = 0.0f;
	for(int i = 0; i < times.size(); i++) {
		mean += times.at(i);
	}
	mean = mean / times.size();
	std::cout << "It took an average of " << mean << " miliseconds, " << 1000.0f/mean << " fps" << std::endl;
	CUDA_CHECK_RETURN(hipHostFree(im));

	return 0;
}
